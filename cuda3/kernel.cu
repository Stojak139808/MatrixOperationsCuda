#include "hip/hip_runtime.h"
﻿//Kacper Stojek 179909
#include <stdio.h>
#include <stdlib.h>

#include "Matrixgpu.h"
#include "Matrix.h"

#include <chrono>
#include <fstream>

void pirnt_textfile(Matrixgpu A, string filename) {

    ofstream out(filename, std::ofstream::out);

    for (int i = 0; i < A.number_of_rows(); ++i) {
        for (int j = 0; j < A.number_of_columns(); ++j) {
            out<<to_string(A.matrix[j][i])<<" ";
        }
        out << "\n";
    }
    
}

void pirnt_textfile(Matrix A, string filename) {

    ofstream out(filename, std::ofstream::out);

    for (int i = 0; i < A.number_of_rows(); ++i) {
        for (int j = 0; j < A.number_of_columns(); ++j) {
            out << to_string(A.matrix[j][i]) << " ";
        }
        out << "\n";
    }

}

void single_test() {

    using std::chrono::high_resolution_clock;
    using std::chrono::duration_cast;
    using std::chrono::duration;
    using std::chrono::milliseconds;

    int mat_size;
    printf("Input A and B size: ");
    scanf("%d", &mat_size);

    float omega = 9;
    float mi = 0;

    Matrixgpu Agpu(mat_size, mat_size);
    Agpu.randomize();
    Agpu.get_values();

    Matrixgpu Bgpu(mat_size, mat_size);
    Bgpu.randomize();
    Bgpu.get_values();

    //copying values
    Matrix Acpu(mat_size, mat_size);
    Matrix Bcpu(mat_size, mat_size);

    for (int i = 0; i < mat_size; ++i) {
        for (int j = 0; j < mat_size; ++j) {
            Acpu.matrix[i][j] = Agpu.matrix[i][j];
            Bcpu.matrix[i][j] = Bgpu.matrix[i][j];
        }
    }
    printf("Calculating C = A*B + A^T * mi + A - B * omega\nStarting test.\n");

    auto t1 = high_resolution_clock::now();
    Matrixgpu Cgpu = Agpu * Bgpu + Agpu.transpose() * mi + Agpu - Bgpu * omega;
    auto t2 = high_resolution_clock::now();

    duration<float, std::milli> gpu_time = t2 - t1;
    printf("Gpu time: %f\n", gpu_time.count());

    auto t3 = high_resolution_clock::now();
    Matrix Ccpu = Acpu * Bcpu + Acpu.transpose() * mi + Acpu - Bcpu * omega;
    auto t4 = high_resolution_clock::now();

    duration<float, std::milli> cpu_time = t4 - t3;
    printf("Cpu time: %f\n%f times faster\n", cpu_time.count(), cpu_time.count() / gpu_time.count());

    Cgpu.get_values();

    pirnt_textfile(Agpu, "A.txt");
    pirnt_textfile(Bgpu, "B.txt");
    pirnt_textfile(Cgpu, "Cgpu.txt");
    pirnt_textfile(Cgpu, "Ccpu.txt");

}

void multiplte_tests() {

    using std::chrono::high_resolution_clock;
    using std::chrono::duration_cast;
    using std::chrono::duration;
    using std::chrono::milliseconds;

    int mat_size = 100;
    float omega = 9;
    float mi = 0;

    printf("n, GPU [ms], CPU [ms], CPU/GPU [ms/ms]\n");

    while (mat_size < 5000) {
        printf("%d ", mat_size);
        Matrixgpu Agpu(mat_size, mat_size);
        Agpu.randomize();
        Agpu.get_values();

        Matrixgpu Bgpu(mat_size, mat_size);
        Bgpu.randomize();
        Bgpu.get_values();

        //copying values
        Matrix Acpu(mat_size, mat_size);
        Matrix Bcpu(mat_size, mat_size);

        for (int i = 0; i < mat_size; ++i) {
            for (int j = 0; j < mat_size; ++j) {
                Acpu.matrix[i][j] = Agpu.matrix[i][j];
                Bcpu.matrix[i][j] = Bgpu.matrix[i][j];
            }
        }

        auto t1 = high_resolution_clock::now();
        Matrixgpu Cgpu = Agpu * Bgpu + Agpu.transpose() * mi + Agpu - Bgpu * omega;
        auto t2 = high_resolution_clock::now();

        duration<float, std::milli> gpu_time = t2 - t1;

        auto t3 = high_resolution_clock::now();
        Matrix Ccpu = Acpu * Bcpu + Acpu.transpose() * mi + Acpu - Bcpu * omega;
        auto t4 = high_resolution_clock::now();

        duration<float, std::milli> cpu_time = t4 - t3;
        printf("%f %f %f\n", cpu_time.count(), gpu_time.count(), cpu_time.count() / gpu_time.count());

        mat_size += 100;
    }

}

__global__ void nullKernel() {

}

int main() {

    nullKernel << <1, 1 >> > ();

    //multiplte_tests();
    
    single_test();

}